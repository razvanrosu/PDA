#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdlib>

// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#define w 6
#define h 6
#define N w*h

__global__ void reduce(int *g_idata, int searchedNumber,  int *ok);
void fill_array(int *a, int n);

int main(void) {
	int a[N];
	int *dev_a;
	int size = N * sizeof(int); // we need space for 512 integers

								// allocate device copies of a, b, c
	hipMalloc((void**)&dev_a, size);

	fill_array(a, N);
	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	int searchedNumber;
	int ok = -1;

	printf("Type a number : ");
	scanf("%d", &searchedNumber);
	dim3 blocksize(6); // create 1D threadblock
	dim3 gridsize(N / blocksize.x);  //create 1D grid

	reduce << <gridsize, blocksize >> > (dev_a, searchedNumber, &ok);
	if (ok != -1) {
		("Found %d on %d position", searchedNumber, ok);
	}
	
	hipFree(dev_a);
	scanf("%d");
	return 0;
}

__global__ void reduce(int *g_idata, int searchedNumber, int *ok) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("%d ", i);

	__syncthreads();
	//printf("%d %d///", g_idata[i], searchedNumber);
	if (g_idata[i] == searchedNumber) {
		printf("Found %d on %d position %d", searchedNumber, i, *ok);
		*ok = i;
	}
}

void fill_array(int *a, int n)
{
	for (int i = 0; i < n; i++)
		a[i] = i;
}
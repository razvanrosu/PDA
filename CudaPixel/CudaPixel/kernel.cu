#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>



#include "hip/hip_runtime_api.h"
#include <iostream>  
#include <fstream>   
#include <iomanip>   
#include <windows.h>
#include <io.h>                  
#include <stdio.h>
#include<conio.h>
#include <cstdlib>
#include "cstdlib"
#include <process.h>
#include <stdlib.h>
#include <malloc.h>
#include <ctime>
using namespace std;

#define MEDIAN_DIMENSION  3 // For matrix of 3 x 3. We can Use 5 x 5 , 7 x 7 , 9 x 9......   
#define MEDIAN_LENGTH 9   // Shoul be  MEDIAN_DIMENSION x MEDIAN_DIMENSION = 3 x 3

#define BLOCK_WIDTH 16  // Should be 8 If matrix is of larger then of 5 x 5 elese error occur as " uses too much shared data "  at surround[BLOCK_WIDTH*BLOCK_HEIGHT][MEDIAN_LENGTH]
#define BLOCK_HEIGHT 16// Should be 8 If matrix is of larger then of 5 x 5 elese error occur as " uses too much shared data "  at surround[BLOCK_WIDTH*BLOCK_HEIGHT][MEDIAN_LENGTH]

__global__ void MedianFilter_gpu(unsigned short *Device_ImageData, int Image_Width, int Image_Height) {

	__shared__ unsigned short surround[BLOCK_WIDTH*BLOCK_HEIGHT][MEDIAN_LENGTH];

	int iterator;
	const int Half_Of_MEDIAN_LENGTH = (MEDIAN_LENGTH / 2) + 1;
	int StartPoint = MEDIAN_DIMENSION / 2;
	int EndPoint = StartPoint + 1;

	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;

	const int tid = threadIdx.y*blockDim.y + threadIdx.x;

	if (x >= Image_Width || y >= Image_Height)
		return;

	//Fill surround with pixel value of Image in Matrix Pettern of MEDIAN_DIMENSION x MEDIAN_DIMENSION
	if (x == 0 || x == Image_Width - StartPoint || y == 0
		|| y == Image_Height - StartPoint) {
	}
	else {
		iterator = 0;
		for (int r = x - StartPoint; r < x + (EndPoint); r++) {
			for (int c = y - StartPoint; c < y + (EndPoint); c++) {
				surround[tid][iterator] = *(Device_ImageData + (c*Image_Width) + r);
				iterator++;
			}
		}
		//Sort the Surround Array to Find Median. Use Bubble Short  if Matrix oF 3 x 3 Matrix 
		//You can use Insertion commented below to Short Bigger Dimension Matrix  

		////      bubble short //

		for (int i = 0; i<Half_Of_MEDIAN_LENGTH; ++i)
		{
			// Find position of minimum element
			int min = i;
			for (int l = i + 1; l<MEDIAN_LENGTH; ++l)
				if (surround[tid][l] <surround[tid][min])
					min = l;
			// Put found minimum element in its place
			unsigned short  temp = surround[tid][i];
			surround[tid][i] = surround[tid][min];
			surround[tid][min] = temp;
		}//bubble short  end

		 //////insertion sort start   //

		 /*int t,j,i;
		 for ( i = 1 ; i< MEDIAN_LENGTH ; i++) {
		 j = i;
		 while ( j > 0 && surround[tid][j] < surround[tid][j-1]) {
		 t= surround[tid][j];
		 surround[tid][j]= surround[tid][j-1];
		 surround[tid][j-1] = t;
		 j--;
		 }
		 }*/

		 ////insertion sort end   



		*(Device_ImageData + (y*Image_Width) + x) = surround[tid][Half_Of_MEDIAN_LENGTH - 1];   // it will give value of surround[tid][4] as Median Value if use 3 x 3 matrix
		__syncthreads();
	}
}

int main(int argc, const char** argv)
{
	int dataLength;
	int p1;
	unsigned short* Host_ImageData = NULL;
	ifstream is; // Read File 
	is.open("maxresdefault", ios::binary);

	// get length of file:
	is.seekg(0, ios::end);
	dataLength = is.tellg();
	is.seekg(0, ios::beg);

	Host_ImageData = new  unsigned short[dataLength * sizeof(char) / sizeof(unsigned short)];
	is.read((char*)Host_ImageData, dataLength);
	is.close();

	int Image_Width = 1580;
	int Image_Height = 1050;

	unsigned short *Host_ResultData = (unsigned short *)malloc(dataLength);
	unsigned short *Device_ImageData = NULL;

	/////////////////////////////
	// As First time hipMalloc take more time  for memory alocation, i dont want to cosider this time in my process. 
	//So Please Ignore Code For Displaying First CudaMelloc Time
	clock_t begin = clock();
	unsigned short *forFirstCudaMalloc = NULL;
	hipMalloc((void**)&forFirstCudaMalloc, dataLength * sizeof(unsigned short));
	clock_t end = clock();
	double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
	cout << "First CudaMelloc time = " << elapsed_secs << "  Second\n";
	hipFree(forFirstCudaMalloc);
	////////////////////////////

	//Actual Process Starts From Here 
	clock_t beginOverAll = clock();   //
	hipMalloc((void**)&Device_ImageData, dataLength * sizeof(unsigned short));
	hipMemcpy(Device_ImageData, Host_ImageData, dataLength, hipMemcpyHostToDevice);// copying Host Data To Device Memory For Filtering

	int x = static_cast<int>(ceilf(static_cast<float>(1580.0) / BLOCK_WIDTH));
	int y = static_cast<int>(ceilf(static_cast<float>(1050.0) / BLOCK_HEIGHT));

	const dim3 grid(x, y, 1);
	const dim3 block(BLOCK_WIDTH, BLOCK_HEIGHT, 1);

	begin = clock();

	MedianFilter_gpu << <grid, block >> >(Device_ImageData, Image_Width, Image_Height);
	hipDeviceSynchronize();

	end = clock();
	elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
	cout << "Process time = " << elapsed_secs << "  Second\n";

	hipMemcpy(Host_ResultData, Device_ImageData, dataLength, hipMemcpyDeviceToHost); // copying Back Device Data To Host Memory To write In file After Filter Done

	clock_t endOverall = clock();
	elapsed_secs = double(endOverall - beginOverAll) / CLOCKS_PER_SEC;
	cout << "Complete Time  = " << elapsed_secs << "  Second\n";

	ofstream of2;   //Write Filtered Image Into File
	of2.open("D:\\Filtered_Image.raw", ios::binary);
	of2.write((char*)Host_ResultData, dataLength);
	of2.close();
	cout << "\nEnd of Writing File.  Press Any Key To Exit..!!";
	hipFree(Device_ImageData);
	delete Host_ImageData;
	delete Host_ResultData;

	getch();
	return 0;
}


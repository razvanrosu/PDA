
#include "hip/hip_runtime.h"


#include <stdio.h>#
#include <iostream>

static unsigned char *texels;
static int width, height;

static void readBmp(char *filename)
{
	FILE *fd;
	fd = fopen(filename, "rb");
	if (fd == NULL)
	{
		printf("Error: fopen failed\n");
		return;
	}

	unsigned char header[54];

	// Read header
	fread(header, sizeof(unsigned char), 54, fd);

	// Capture dimensions
	width = *(int*)&header[18];
	height = *(int*)&header[22];

	int padding = 0;

	// Calculate padding
	while ((width * 3 + padding) % 4 != 0)
	{
		padding++;
	}

	// Compute new width, which includes padding
	int widthnew = width * 3 + padding;

	// Allocate memory to store image data (non-padded)
	texels = (unsigned char *)malloc(width * height * 3 * sizeof(unsigned char));
	if (texels == NULL)
	{
		printf("Error: Malloc failed\n");
		return;
	}

	// Allocate temporary memory to read widthnew size of data
	unsigned char* data = (unsigned char *)malloc(widthnew * sizeof(unsigned int));

	// Read row by row of data and remove padded data.
	for (int i = 0; i<height; i++)
	{
		// Read widthnew length of data
		fread(data, sizeof(unsigned char), widthnew, fd);

		// Retain width length of data, and swizzle RB component.
		// BMP stores in BGR format, my usecase needs RGB format
		for (int j = 0; j < width * 3; j += 3)
		{
			int index = (i * width * 3) + (j);
			texels[index + 0] = data[j + 2];
			texels[index + 1] = data[j + 1];
			texels[index + 2] = data[j + 0];
		}
	}

	free(data);
	fclose(fd);
}





hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
	printf("Start reading image\n");
	readBmp("cEdDG.bmp");
	printf("Finished");
	printf("%s", texels);
	scanf("%d");
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.

